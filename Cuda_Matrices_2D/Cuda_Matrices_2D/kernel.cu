#include "hip/hip_runtime.h"
#include ""
#include "Matrix.h"

#include <stdio.h>

int main()
{  
    Matrix* A = new Matrix(3,3);
    A->aleatorizarRango(0, 10);
    A->print();

    Matrix* B = new Matrix(3, 2);
    B->aleatorizarRango(0,10);
    B->print();

    float* d_A = nullptr;
    float* d_B = nullptr;
    float* d_C = nullptr;
    size_t pitch_A;
    size_t pitch_B;
    size_t pitch_C;

    if (hipMallocPitch((void**)&d_A, &pitch_A, sizeof(float)*A->columnas, A->filas) != hipSuccess)
        std::cout << "Error al crear memoria (A)." << std::endl;
    if (hipMallocPitch((void**)&d_B, &pitch_B, sizeof(float) * B->columnas, B->filas) != hipSuccess)
        std::cout << "Error al crear memoria (B)." << std::endl;

    if(hipMemcpy2D(d_A, pitch_A, A->datos, sizeof(float) * A->columnas, sizeof(float) * A->columnas, A->filas, hipMemcpyHostToDevice) != hipSuccess)
        std::cout << "Error al copiar en memoria (A)." << std::endl;
    if (hipMemcpy2D(d_B, pitch_B, B->datos, sizeof(float) * B->columnas, sizeof(float) * B->columnas, B->filas, hipMemcpyHostToDevice) != hipSuccess)
        std::cout << "Error al copiar en memoria (B)." << std::endl;

    Matrix* D = new Matrix(3, 2);
    if (hipMemcpy2D(D->datos, sizeof(float) * D->columnas, d_B, pitch_B, sizeof(float) * B->columnas, B->filas, hipMemcpyDeviceToHost) != hipSuccess)
        std::cout << "Error al copiar en memoria (D)." << std::endl;
    D->print();
    return 0;
}
