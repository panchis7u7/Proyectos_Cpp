#include "hip/hip_runtime.h"
#include ""
#include "Matrix.h"
#include <stdio.h>

int main()
{  
    Matrix<int>* A = new Matrix<int>(13,13);
    A->aleatorizarRango(0, 10);
    A->print();

    Matrix<int>* B = new Matrix<int>(3, 2);
    B->aleatorizarRango(0,10);
    B->print();

    float* d_A = nullptr;
    float* d_B = nullptr;
    size_t pitch_A;
    size_t pitch_B;

    if (hipMallocPitch((void**)&d_A, &pitch_A, sizeof(float)*A->columnas, A->filas) != hipSuccess)
        std::cout << "Error al crear memoria (A)." << std::endl;
    if (hipMallocPitch((void**)&d_B, &pitch_B, sizeof(float) * B->columnas, B->filas) != hipSuccess)
        std::cout << "Error al crear memoria (B)." << std::endl;

    if(hipMemcpy2D(d_A, pitch_A, A->datos, sizeof(float) * A->columnas, sizeof(float) * A->columnas, A->filas, hipMemcpyHostToDevice) != hipSuccess)
        std::cout << "Error al copiar en memoria (A)." << std::endl;
    if (hipMemcpy2D(d_B, pitch_B, B->datos, sizeof(float) * B->columnas, sizeof(float) * B->columnas, B->filas, hipMemcpyHostToDevice) != hipSuccess)
        std::cout << "Error al copiar en memoria (B)." << std::endl;

    Matrix<int>* D = new Matrix<int>(A->filas, A->columnas);
    if (hipMemcpy2D(D->datos, sizeof(float) * D->columnas, d_A, pitch_A, sizeof(float) * A->columnas, A->filas, hipMemcpyDeviceToHost) != hipSuccess)
        std::cout << "Error al copiar en memoria (D)." << std::endl;
    D->print();
    return 0;
}
