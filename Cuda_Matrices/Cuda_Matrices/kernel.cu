#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>

using std::cout;
using std::generate;
using std::vector;

class Matrix {
public:
    Matrix(int filas, int columnas)
    {
        this->filas = filas;
        this->columnas = columnas;
        this->data = new int* [filas];
        for (int i = 0; i < filas; i++)
        {
            this->data[i] = new int[columnas];
        }
    }

    void print() {
        for (int i = 0; i < this->filas; i++)
        {
            std::cout << "|";
            for (int j = 0; j < this->columnas; j++)
            {
                std::cout << " " << this->data[i][j] << " ";
            }
            std::cout << "|";
            std::cout << std::endl;
        }
        std::cout << std::endl;
    }

    void Matrix::aleatorizar() {
        for (size_t i = 0; i < this->filas; i++)
        {
            for (size_t j = 0; j < this->columnas; j++)
            {
                //Genera numero aleatorio entre -1 y 1
                this->data[i][j] = (-1) + static_cast <int> (rand()) / (static_cast <int> (RAND_MAX / (1 - (-1))));
            }
        }
    }

    void Matrix::aleatorizarRango(int rango1, int rango2) {
        for (size_t i = 0; i < this->filas; i++)
        {
            for (size_t j = 0; j < this->columnas; j++)
            {
                //Genera numero aleatorio entre -1 y 1
                this->data[i][j] = (rango1) + static_cast <int> (rand()) / (static_cast <int> (RAND_MAX / (rango2 - (-1))));
            }
        }
    }

    int* Matrix::toVector() {
        int* res = new int[this->filas * this->columnas];
        int k = 0;
        for (int i = 0; i < this->filas; i++)
        {
            for (int j = 0; j < this->columnas; j++)
            {
                res[k] = this->data[i][j];
                k++;
            }
        }
        return res;
    }

    static Matrix* Matrix::toMatrix(int* matrix, int filas, int columnas) {
        Matrix* resultado = new Matrix(filas, columnas);
        int k = 0;
        for (int i = 0; i < filas; i++)
        {
            for (int j = 0; j < columnas; j++)
            {
                resultado->data[i][j] = matrix[k];
                k++;
            }
        }
        return resultado;
    }

    static Matrix* Matrix::multiplicar(Matrix* A, Matrix* B) {
        Matrix* resultado = new Matrix(A->filas, B->columnas);
        int suma = 0;
        for (short i = 0; i < resultado->filas; i++)
        {
            for (short j = 0; j < resultado->columnas; j++)
            {
                suma = 0;
                for (short k = 0; k < A->columnas; k++)
                {
                    suma += A->data[i][k] * B->data[k][j];
                }
                resultado->data[i][j] = suma;
            }
        }
        return resultado;
    }

    int** data;
    int filas, columnas;
private:
};

__global__ void matrixMul(int* A, int* B, int* C, 
                          int aF, int aC,
                          int bF, int bC,
                          int cF, int cC) {
    // Compute each thread's global row and column index
    int row = (blockIdx.y * blockDim.y) + threadIdx.y;
    int col = (blockIdx.x * blockDim.x) + threadIdx.x;

    // Iterate over row, and down column
    ////c[row * N + col] = 0;
    if (aC != bF) return;
    if ((row < aF) && (col < bC)) {
        for (int k = 0; k < aC; ++k) {
            // Accumulate results for a single element
            C[row * cC + col] += A[row * aC + k] * B[k * bC + col];
        }
    }
    //C[row * aF + col] = 0;
}

int main()
{
    srand(time(NULL));
    //Matriz A
    Matrix* A = new Matrix(25, 25);
    A->aleatorizarRango(0, 20);
    A->print();
    int* h_A = A->toVector();

    //Matriz B
    Matrix* B = new Matrix(25, 20);
    B->aleatorizarRango(0, 20);
    B->print();
    int* h_B = B->toVector();

    //Matrix* C = Matrix::multiplicar(A, B);
    //C->print();
    Matrix* C = new Matrix(A->filas, B->columnas);
    C->aleatorizarRango(0, 20);
    C->print();
    int* h_C = C->toVector();

    Matrix* res = Matrix::multiplicar(A,B);
    res->print();

    int* d_A;
    int sizeA = sizeof(int) * A->filas * A->columnas;
    int* d_B;
    int sizeB = sizeof(int) * B->filas * B->columnas;
    int* d_C;
    int sizeC = sizeof(int) * C->filas * C->columnas;

    // Allocate device memory
    if(hipMalloc((void**)&d_A, sizeA) != hipSuccess)
        std::cout << "Error al despachar A en memoria." << std::endl;
    if(hipMalloc((void**)&d_B, sizeB) != hipSuccess)
        std::cout << "Error al despachar B en memoria." << std::endl;
    if(hipMalloc((void**)&d_C, sizeC) != hipSuccess)
        std::cout << "Error al despachar C en memoria." << std::endl;

    // Copy data to the device
    if(hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice) != hipSuccess)
        std::cout << "Error en MemCpy (A)." << std::endl;
    if(hipMemcpy(d_B, h_B, sizeB, hipMemcpyHostToDevice) != hipSuccess)
        std::cout << "Error en MemCpy (B)." << std::endl;

    // Threads per CTA dimension
    //int THREADS = 32;

    // Blocks per grid dimension (assumes THREADS divides N evenly)
    //int BLOCKS = N / THREADS;
    int BLOCKS = 1;

    // Use dim3 structs for block and grid dimensions
    dim3 threads(C->columnas, C->filas);
    //dim3 blocks(A->filas, B->columnas);

    // Launch kernel
    matrixMul << <1, threads >> > (d_A, d_B, d_C, A->filas, A->columnas, B->filas, B->columnas, C->filas, C->columnas);

    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // Copy back to the host
    if(hipMemcpy(h_C, d_C, sizeC, hipMemcpyDeviceToHost) != hipSuccess)
        std::cout << "Error en MemCpy (C)." << std::endl;
    Matrix* D = Matrix::toMatrix(h_C, C->filas, C->columnas);
    D->print();

    cout << "COMPLETED SUCCESSFULLY\n";

    // Free memory on device
    Error:
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}