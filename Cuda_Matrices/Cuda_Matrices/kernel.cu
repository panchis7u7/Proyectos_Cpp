#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>

using std::cout;
using std::generate;
using std::vector;

class Matrix {
public:
    Matrix(int filas, int columnas)
    {
        this->filas = filas;
        this->columnas = columnas;
        this->data = new int* [filas];
        for (int i = 0; i < filas; i++)
        {
            this->data[i] = new int[columnas];
        }
    }

    void print() {
        for (int i = 0; i < this->filas; i++)
        {
            std::cout << "|";
            for (int j = 0; j < this->columnas; j++)
            {
                std::cout << " " << this->data[i][j] << " ";
            }
            std::cout << "|";
            std::cout << std::endl;
        }
        std::cout << std::endl;
    }

    void Matrix::aleatorizar() {
        for (size_t i = 0; i < this->filas; i++)
        {
            for (size_t j = 0; j < this->columnas; j++)
            {
                //Genera numero aleatorio entre -1 y 1
                this->data[i][j] = (-1) + static_cast <int> (rand()) / (static_cast <int> (RAND_MAX / (1 - (-1))));
            }
        }
    }

    void Matrix::aleatorizarRango(int rango1, int rango2) {
        for (size_t i = 0; i < this->filas; i++)
        {
            for (size_t j = 0; j < this->columnas; j++)
            {
                //Genera numero aleatorio entre -1 y 1
                this->data[i][j] = (rango1) + static_cast <int> (rand()) / (static_cast <int> (RAND_MAX / (rango2 - (-1))));
            }
        }
    }

    static Matrix* Matrix::multiplicar(Matrix* A, Matrix* B) {
        Matrix* resultado = new Matrix(A->filas, B->columnas);
        int suma = 0;
        for (short i = 0; i < resultado->filas; i++)
        {
            for (short j = 0; j < resultado->columnas; j++)
            {
                suma = 0;
                for (short k = 0; k < A->columnas; k++)
                {
                    suma += A->data[i][k] * B->data[k][j];
                }
                resultado->data[i][j] = suma;
            }
        }
        return resultado;
    }
    int** data;
    int filas, columnas;
private:
};

__global__ void matrixMul(const Matrix* A, const Matrix* B, Matrix* C) {
    // Compute each thread's global row and column index
    int col = (blockIdx.y * blockDim.y) + threadIdx.y;
    int row = (blockIdx.x * blockDim.x) + threadIdx.x;

    // Iterate over row, and down column
    //c[row * N + col] = 0;
    for (int k = 0; k < A->filas; k++) {
        // Accumulate results for a single element
        //c[row * N + col] += a[row * N + k] * b[k * N + col];
        //C->data[row][col] += A->data[row][k] * B->data[k][row];
        C->data[row][col] = 0;
    }
}

int main()
{
    srand(time(NULL));
    //Matriz A
    Matrix* A = new Matrix(3, 3);
    A->aleatorizarRango(0, 20);
    A->print();

    //Matriz B
    Matrix* B = new Matrix(3, 2);
    B->aleatorizarRango(0, 20);
    B->print();

    //Matrix* C = Matrix::multiplicar(A, B);
    //C->print();
    Matrix* C = new Matrix(A->filas, B->columnas);
    C->aleatorizar();
    C->print();

    Matrix* d_A = A;//(Matrix*)malloc(sizeof(Matrix));
    Matrix* d_B = B;
    Matrix* d_C = C;

    // Allocate device memory
    hipMalloc(&d_A, sizeof(Matrix));
    hipMalloc(&d_B, sizeof(Matrix));

    // Copy data to the device
    hipMemcpy(&d_A, &A, sizeof(Matrix), hipMemcpyHostToDevice);
    hipMemcpy(&d_B, &B, sizeof(Matrix), hipMemcpyHostToDevice);

    // Threads per CTA dimension
    //int THREADS = 32;

    // Blocks per grid dimension (assumes THREADS divides N evenly)
    //int BLOCKS = N / THREADS;
    int BLOCKS = 1;

    // Use dim3 structs for block and grid dimensions
    dim3 threads(A->filas, B->columnas);
    dim3 blocks(BLOCKS);

    // Launch kernel
    matrixMul << <blocks, threads >> > (d_A, d_B, d_C);

    // Copy back to the host
    hipMemcpy(&C, &d_C, sizeof(Matrix), hipMemcpyDeviceToHost);
    C->print();
    // Check result
    //verify_result(h_a, h_b, h_c, N);
    //print_result(h_a, h_b, h_c, N);

    cout << "COMPLETED SUCCESSFULLY\n";

    // Free memory on device
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}